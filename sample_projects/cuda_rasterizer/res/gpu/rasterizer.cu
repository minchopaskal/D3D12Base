#include "hip/hip_runtime.h"
// Includes that fix syntax highlighting
#ifdef D3D12_DEBUG
#include ""
#include "stdio.h"
#include "math_functions.h"
#endif

#include "hip/hip_math_constants.h"
#include "cuda_cpu_common.h"
#include "rasterizer_utils.cuh"

typedef Vertex(*VSShader)(const Vertex*);
typedef float4(*PSShader)(const Vertex*);

extern "C" {
	__device__ VSShader vsShader;
	__device__ PSShader psShader;

	/// Pointers to UAV resources
	cfloat *depthBuffer;
	cfloat *renderTarget;
	cint *pixelsBarriers;
	__device__ Vertex *vertexBuffer;
	cuint *indexBuffer;
	cvoid *resources[MAX_RESOURCES_COUNT];
	cfloat clearColor[4];
	cbool useDepthBuffer;
	__constant__ CudaRasterizerCullType cullType;
	cuint width;
	cuint height;

	FORCEINLINE dfloat getDiscreetValue(const float value, const unsigned int steps) {
		return (int)min(max((value + 1.f) * 0.5f * steps, 0.f), float(steps));
	}

	FORCEINLINE dfloat2 getDiscreeteCoordinates(const float4 p, const unsigned int width, const unsigned int height) {
		float2 result;
		// [-1.f, 1.f] -> [0; width/height]
		result.x = getDiscreetValue(p.x, width);
		result.y = getDiscreetValue(p.y, height);

		return result;
	}

	FORCEINLINE dfloat3 findBarys(
		const float2 p0,
		const float2 p1,
		const float2 p2,
		const int2 p
	) {
		// TODO: Crammer's rule but compacted. Test perf with classic
		float3 u = cross(
			make_float3(p1.x - p0.x, p2.x - p0.x, p0.x - p.x),
			make_float3(p1.y - p0.y, p2.y - p0.y, p0.y - p.y)
		);

		if (fabs(u.z) < 1.f) {
			return make_float3(-1.f, 1.f, 1.f);
		}

		float3 res;
		res.y = u.x / u.z;
		res.z = u.y / u.z;
		res.x = 1.f - (res.y + res.z);
		return res;
	}

	/// Computes bounding box of a triangle given its coordinates.
	/// Bounding box layout:
	/// float4(bbox.topLeftXCoordinate, bbox.topLeftYCoordinate, bbox.height, bbox.width)
	dfloat4 computeBoundingBox(
		const float2 pos0,
		const float2 pos1,
		const float2 pos2
	) {
		float4 result;
		result.x = min(min(pos0.x, pos1.x), pos2.x);
		result.y = min(min(pos0.y, pos1.y), pos2.y);
		result.z = (int)max(max(pos0.y, pos1.y), pos2.y) - result.y;
		result.w = (int)max(max(pos0.x, pos1.x), pos2.x) - result.x;

		return result;
	}

	FORCEINLINE __device__ Vertex getInterpolatedVertex(float3 barys, Vertex v0, Vertex v1, Vertex v2) {
		Vertex result;
		result.position = barys.x * v0.position + barys.y * v1.position + barys.z * v2.position;
		/*result.normal = barys.x * v0.normal + barys.y * v1.normal + barys.z * v2.normal;
		result.uv = barys.x * v0.uv + barys.y * v1.uv + barys.z * v2.uv;*/

		return result;
	}

	// Rasterization functions
	gvoid shadeTriangle(
		const unsigned int primitiveID,
		const unsigned int numPrimitives,
		const float4 bbox,
		const Vertex v0,
		const Vertex v1,
		const Vertex v2,
		const float2 dp0, // screen-space coordinates of the vertices. to-do: change name
		const float2 dp1, // screen-space coordinates of the vertices. to-do: change name
		const float2 dp2, // screen-space coordinates of the vertices. to-do: change name
		const float3 edge0,
		const float3 edge1,
		const float3 edge2,
		const unsigned int width,
		const unsigned int height
	) {
		const unsigned int threadID = blockIdx.x * blockDim.x + threadIdx.x;
		const unsigned int stride = gridDim.x * blockDim.x;
		const unsigned int pixelsInBBox = bbox.z * bbox.w;

		const unsigned int bboxWidth = bbox.w;

		for (int i = threadID; i < pixelsInBBox; i += stride) {
			int2 p;
			p.x = threadID % bboxWidth + bbox.x;
			p.y = threadID / bboxWidth + bbox.y;

			const float dX = p.x - bbox.x;
			const float dY = p.y - bbox.y;
			const float edge0Eq = edge0.z + dX * edge0.y - dY * edge0.x;
			const float edge1Eq = edge1.z + dX * edge1.y - dY * edge1.x;
			const float edge2Eq = edge2.z + dX * edge2.y - dY * edge2.x;

			// Check if point is inside the triangle by checking it agains the
			// edge equations of the triangle edges.
			if (edge0Eq > 0 || edge1Eq > 0 || edge2Eq > 0) {
				continue;
			}

			const unsigned int y = height - p.y - 1;
			const unsigned int pixelIndex = (y * width + p.x);
			const unsigned int pixelIndexOffset = pixelIndex * 4;

			assert(pixelIndex < width * height);

			// TODO: do not do the culling here
			if (pixelIndex >= width * height * 4) {
				return;
			}

			// Transform vertex NDC coordinates to screen coords and
			// find barys of shaded pixel based on that
			const float3 barys = findBarys(dp0, dp1, dp2, p);
			Vertex interpolatedVertex = getInterpolatedVertex(barys, v0, v1, v2);

			bool passDepthTest = true;
			while (true) {
				if (!useDepthBuffer) {
					break;
				}

				passDepthTest = false;
				float oldZ = depthBuffer[pixelIndex];
				if (oldZ < interpolatedVertex.position.z) {
					break;
				}

				passDepthTest = true;
				if (atomicCAS(
						(unsigned int*)&depthBuffer[pixelIndex],
						__float_as_uint(oldZ),
						__float_as_uint(interpolatedVertex.position.z)) == __float_as_uint(oldZ)) {
					break;
				}
			}

			if (!passDepthTest) {
				continue;
			}

			float4 color = psShader(&interpolatedVertex);
			renderTarget[pixelIndexOffset + 0] = color.x;
			renderTarget[pixelIndexOffset + 1] = color.y;
			renderTarget[pixelIndexOffset + 2] = color.z;
			renderTarget[pixelIndexOffset + 3] = color.w;
		}
	}

	gvoid drawIndexed(const int numPrimitives, const unsigned int width, const unsigned int height) {
		const unsigned int primitiveID = blockIdx.x * blockDim.x + threadIdx.x;
		const unsigned int stride = gridDim.x * blockDim.x;

		if (primitiveID >= numPrimitives) {
			return;
		}

		for (int i = primitiveID; i < numPrimitives; i += stride) {
			// 1. RUN VS shader
			Vertex v0 = vsShader(&vertexBuffer[indexBuffer[i * 3 + 0]]);
			Vertex v1 = vsShader(&vertexBuffer[indexBuffer[i * 3 + 1]]);
			Vertex v2 = vsShader(&vertexBuffer[indexBuffer[i * 3 + 2]]);

			// Back-face culling
			// Vertices are now in NDC, so we can test for back-face against
			// the (0, 0, -1) vector which points outside the monitor.
			// TODO: if (cull)
			const float3 ab = fromFloat4(v1.position - v0.position);
			const float3 ac = fromFloat4(v2.position - v0.position);
			const float3 normal = normalize(cross(ac, ab));
			if (cullType != cullType_none) {
				const bool backface = dot(normal, make_float3(0.f, 0.f, -1.f)) < 0;
				if ((cullType == cullType_backface && backface) || (cullType == cullType_frontface && !backface)) {
					continue;
				}
			}

			// TODO: write rasterization in hierarchical approach
			// for each triangle run 1 thread for each 8x8(or smth else) block
			// inside its bounding box. The thread should quickly test if the block 
			// is inside the triangle. Mark the blocks that contain part of the triangle.
			// Second level would be to run threads for each marked block and rasterize
			// the part of the triangle inside.
			
			// 2. FOR EACH TRIANGLE RUN WITH DYNAMIC PARALLELISM 
			// foreach (triangle) // i.e if vertexID % 3 == 0
			//   computeBoundingBox();
			//   numThreads = bbox.width * bbox.height
			//   shadeTriangleKernel<<<~numThreads>>>(triangleIndex, vsOutput)
			// end foreach
			const float2 dp0 = getDiscreeteCoordinates(v0.position, width, height);
			const float2 dp1 = getDiscreeteCoordinates(v1.position, width, height);
			const float2 dp2 = getDiscreeteCoordinates(v2.position, width, height);

			const float2 e0 = dp1 - dp0;
			const float2 e1 = dp2 - dp1;
			const float2 e2 = dp0 - dp2;

			const float4 bbox = computeBoundingBox(dp0, dp1, dp2);
			// Layout of edge vectors:
			// (x coordinate of start of edge in screen-coords, dx, dy, edge equation for top-left bbox corner)
			const float3 edge0 = make_float3(e0.x, e0.y, (bbox.x - dp0.x) * e0.y - (bbox.y - dp0.y) * e0.x);
			const float3 edge1 = make_float3(e1.x, e1.y, (bbox.x - dp1.x) * e1.y - (bbox.y - dp1.y) * e1.x);
			const float3 edge2 = make_float3(e2.x, e2.y, (bbox.x - dp2.x) * e2.y - (bbox.y - dp2.y) * e2.x);

			const unsigned int blockSize = 128;
			const unsigned int numThreads = bbox.z * bbox.w;
			const unsigned int numBlocks = (numThreads / blockSize) + (numThreads % blockSize != 0);

			hipStream_t stream;
			hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
			shadeTriangle<<<numBlocks, blockSize, 0, stream>>>(i, numPrimitives, bbox, v0, v1, v2, dp0, dp1, dp2, edge0, edge1, edge2, width, height);
			hipStreamDestroy(stream);
		}
	}

	gvoid blank(float *target, int width, int height) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		idx = min(idx, width * height - 1) * 4;

		target[idx + 0] = clearColor[0];
		target[idx + 1] = clearColor[1];
		target[idx + 2] = clearColor[2];
		target[idx + 3] = clearColor[3];
	}
}
